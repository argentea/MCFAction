
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <string>
#include <fstream>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <chrono>
#define MAXMY 0x3f3f
#define SIZE 256
#define EDGESIZE 2048
#define DEBUG 1
#define FULLDEBUG 0
using namespace std;
__device__ int kflag;
__device__ int tans;
__device__ int minRise;
__device__ int kpushListPo[SIZE];
__device__ int kpushListNa[SIZE];
__device__ bool knodesRisePrice[SIZE];
__device__ void kcheck(
		const int* kg,
		int lnodes,
		int rnodes
		){
	for(int i = lnodes; i < rnodes; i++){
		if(kg[i] != 0){
			atomicAnd(&kflag, 0);
		}
	}
	__syncthreads();
}
__device__ void printNodes(const int* nodes, int numNodes, const char* name){
	printf("*******************\n");
	printf(name);
	for(int i = 0; i < numNodes; i++){
		printf("%d\t", nodes[i]);
	}
	printf("\n*******************\n");
}
__device__ void printGraph(const int* graph, int numNodes, const char* name){
	printf("*******************\n");
	printf("%d", numNodes);
	printf(name);
//	for(int i = 0; i < numNodes; i++){
//		for(int j = 0; j < numNodes; j++){
//			printf("%d\t", graph[i*numNodes + j]);
//		}
//		printf("\n");
//	}
	printf("*******************\n");
}

__device__ unsigned int justForTest = 0;
__device__ void dcostScalingInit(
		const int costScale,
		const int gdelta,
		const int C,
		const int ledges,
		const int redges,
		const int lnodes,
		const int rnodes,
		const int knumNodes,
		const int* edges,
		const int* costRaw,
		int* cost,
		int* price){
	int ti,tj;
	for(int i = ledges; i < redges; i++){
		ti = edges[i*2 + 0];
		tj = edges[i*2 + 1];
		if(costRaw[ti*knumNodes + tj] <= C){
			cost[ti*knumNodes + tj] = costRaw[ti*knumNodes + tj]/(1 << costScale);
		}
	}
	for(int i = lnodes; i < rnodes; i++){
		price[i]*=(1 << gdelta);
	}
	return;
}
__device__ void pushFlow(
		const int lnodes,
		const int rnodes,
		const int ledges,
		const int redges,
		const int epsilon,
		const int knumNodes,
		int* kflow,
		const int* krb,
		const int* klb,
		const int* kprice,
		const int* kcost,
		const int* kedges,
		int* kg
		){
	for(int i = lnodes; i < rnodes; i++){
		kpushListPo[i] = -1;
		kpushListNa[i] = -1;
	}
	for(int i = ledges; i < redges; i++){
		int ti,tj;
		ti = kedges[i*knumNodes + 0];
		tj = kedges[i*knumNodes + 1];
		if(kcost[ti*knumNodes + tj] - kprice[ti] + kprice[tj] + epsilon == 0&&kg[ti] >0){
			atomicExch(kpushListPo + ti, tj);
			continue;
		}
		if(kcost[ti*knumNodes + tj] - kprice[ti] + kprice[tj] - epsilon == 0&&kg[tj] > 0){
			atomicExch(kpushListNa + tj, ti);
			continue;
		}
	}
	int delta;
	for(int i = lnodes; i < rnodes; i++){
		if(kpushListPo[i] != -1){
//			delta = min(kg[i], krb[i*knumNodes + kpushListPo[i]] - kflow[i*knumNodes + kpushListPo[i]]);
			kflow[i*knumNodes + kpushListPo[i]] += delta;
			atomicSub(kg+i, delta);
			atomicSub(kg + kpushListPo[i], delta);
		}
	}
	for(int i = lnodes; i < rnodes; i++){
		if(kpushListNa[i] != -1){
//			delta = min(kg[i], kflow[kpushListNa[i]*knumNodes + i]);
			kflow[kpushListNa[i]*knumNodes + i] -= delta;
			atomicSub(kg+i, delta);
			atomicSub(kg + kpushListNa[i], delta);
		}
	}
	return ;
}
__device__ void priceRise(
		const int lnodes,
		const int rnodes,
		const int ledges,
		const int redges,
		const int epsilon,
		const int knumNodes,
		const int* kflow,
		const int* krb,
		const int* klb,
		const int* kprice,
		const int* kcost,
		const int* edges,
		const int* kg
		){
	int ti,tj,swap,tmpa,tmpb;
	for(int i = lnodes; i < rnodes; i++){
		if(kg[i] > 0){
			knodesRisePrice[i] = true;
		}else {
			knodesRisePrice[i] = false;
		}
	}
	for(int i = ledges; i < redges; i++){
		ti = edges[i*2 + 0];
		tj = edges[i*2 + 1];
		if(knodesRisePrice[ti]!=knodesRisePrice[tj]){
			if(knodesRisePrice[tj]){
				swap  = ti;
				ti = tj;
				tj = swap;
			}
			if(kflow[ti*knumNodes + tj] < krb[ti*knumNodes + tj]){
				tmpb = kprice[tj] + kcost[ti*knumNodes + tj] + epsilon - kprice[ti];
				if(tmpb >= 0){
					atomicMax(&minRise, tmpb);
				}
			}
			if(kflow[tj*knumNodes + ti] > klb[tj*knumNodes + ti]){
				tmpa = kprice[tj] - kcost[tj*knumNodes + ti] + epsilon - kprice[ti];
				if(tmpa > 0){
					atomicMax(&minRise, tmpa);
				}
			}
		}
	}
}
__global__ void __launch_bounds__(1024, 1)
auction_algorithm_kernel(
		const int knumNodes,
		const int knumEdges,
		const int kthreadNum,
		const int kC,

		const int* kedges,
		int* kcost,
		const int* kcostRaw,
		int* kg,
		const int* kgraw,
		const int* klb,
		const int* krb,
		int* kprice,
		int* kflow){
	const int threadId = threadIdx.x;

	
	if(threadId == 0){
		printf("in kernel\n");
	}
	__syncthreads();


	int kepsilon = 1;
	int edgesDivThread;
	int edgesModThread;
	//[edgesl,edgesr) is the range of edges that the thread produre
	int ledges;
	int redges;

	int nodesDivThread;
	int nodesModThread;
	int lnodes;
	int rnodes;

	int totalIteratorNum = 0;
	int iteratorNum = 0;
	int allIterater = 0;
	int tmpa = 0;
	int tmpb = 0;
	int tmpi = 0;
	int scalingFactor = 2;
	int costScale = 9;
	int gdelta = 0;

	int kti;
	int ktj;

	edgesDivThread = knumEdges/kthreadNum;
	edgesModThread = knumEdges%kthreadNum;
	
	if(threadId < edgesModThread){
		ledges = threadId*(edgesDivThread + 1);
		redges = (threadId + 1)*(edgesDivThread + 1);
	}else {
		ledges = threadId*edgesDivThread + edgesModThread;
		redges = (threadId + 1)*edgesDivThread + edgesModThread;
	}
	
	nodesDivThread = knumNodes/kthreadNum;
	nodesModThread = knumNodes%kthreadNum;

	if(threadId < nodesModThread){
		lnodes = threadId*(nodesDivThread + 1);
		rnodes = (threadId + 1)*(nodesDivThread + 1);
	}else{
		lnodes = threadId*nodesDivThread + nodesModThread;
		rnodes = (threadId + 1)*nodesDivThread + nodesModThread;
	}
#if FULLDEBUG
	printf("threadId: %d, ledges: %d, redges: %d\n", threadId, ledges, redges);
	__syncthreads();
	for(int i = ledges; i < redges; i++){
		kflow[kedges[i*2 + 0] * knumNodes + kedges[i*2 + 1]] = atomicAdd(&justForTest, 1);
		printf("%d\n", kflow[kedges[i*2 + 0] * knumNodes + kedges[i*2 + 1]]);
	}
	__syncthreads();
#endif
	while(costScale >= 0){

		int ktmp = 1<<costScale;

		for(int i = ledges; i < redges; i++){
			kti = kedges[i*2 + 0];
			ktj = kedges[i*2 + 1];
			kflow[kti * knumNodes + ktj] = 0;
			if(kcostRaw[kti*knumNodes + ktj] <= kC){
				kcost[kti*knumNodes + ktj] = kcostRaw[kti*knumNodes + ktj]/ktmp;
			}
		}
		for(int i = lnodes; i < rnodes; i++){
			kprice[i]*=(1 << gdelta);
		}
		__syncthreads();
		for(int i = ledges; i < redges; i++){
				kti = kedges[i*2 + 0];
				ktj = kedges[i*2 + 1];
				if(kcost[kti*knumNodes+ktj] - kprice[kti] + kprice[ktj] + kepsilon <= 0){
					kg[kti] -= krb[kti*knumNodes+ktj];
					kg[ktj] += krb[kti*knumNodes+ktj];
					kflow[kti*knumNodes+ktj] = krb[kti*knumNodes+ktj];
				}
		}
		iteratorNum = 0;
		if(threadId == 0)
		{
			kflag = true;
		}
#if FULLDEBUG
		if(threadId == 0){
			printNodes(kg, knumNodes, "g");
		}
		__syncthreads();

		for(int i = lnodes; i < rnodes; i++){
			kg[i] = 0;
		}
		kcheck(
				kg,
				lnodes,
				rnodes
			  );
		__syncthreads();
		if(threadId == 0){
			printf("\nkflag should be true: %d\n", kflag);
			kg[knumNodes/2] = 1;
			printNodes(kg, knumNodes, "g");
		}
		__syncthreads();
		kcheck(
				kg,
				lnodes,
				rnodes
			  );
		__syncthreads();
		if(threadId == 0)
			printf("\nkflag should be false: %d\n", kflag);
		break;
#endif
		__syncthreads();
		kcheck(
			kg,
			lnodes,
			rnodes
		);
		__syncthreads();

		while(!kflag){
			tmpb = 0;
/*			pushFlow(
				lnodes,
				rnodes,
				ledges,
				redges,
				kepsilon,
				knumNodes,
				kflow,
				krb,
				klb,
				kprice,
				kcost,
				kedges,
				kg
				);*/
			if(threadId == 0){
				minRise = MAXMY;
			}
			__syncthreads();
/*			priceRise(
				lnodes,
				rnodes,
				ledges,
				redges,
				kepsilon,
				knumNodes,
				kflow,
				krb,
				klb,
				kprice,
				kcost,
				kedges,
				kg
				);
*/
			__syncthreads();
			if(threadId == 0){
				if(minRise == MAXMY){
					minRise = 0;
				}
			}
			__syncthreads();
			for(int i = lnodes; i < rnodes; i++){
				if(knodesRisePrice[i]){
					kprice[i] += minRise;
				}
			}
			__syncthreads();
			iteratorNum++;
			totalIteratorNum++;
			if(iteratorNum == 5){
				break;
			}

			if(threadId == 0)
			{
				kflag = true;
			}
			kcheck(
				kg,
				lnodes,
				rnodes
			);
			__syncthreads();
		}
		int tans = 0;
		for(int i = ledges; i < redges; i++){
			kti = kedges[i*2 + 0];
			ktj = kedges[i*2 + 1];
			atomicAdd(&tans, kflow[kti*knumNodes + ktj]*kcostRaw[kti*knumNodes + ktj]);
		}
		if(threadId == 0){
			printf("temporary ans: %d\n",tans);
		}
		if(costScale ==0){
			break;
		}
		gdelta = costScale - max(0, costScale - scalingFactor);
		costScale = max(0, costScale - scalingFactor);
	}


	if(threadId == 0)
	{
		printGraph(kcost, knumNodes,"cost");
		printf("kenerl end\n");
	}
	__syncthreads();
}

void run_auction(
		int numNodes,
		int numEdges,
		int threadNum,
		int dC,

		int* hedges,
		int* hcost,
		int* hg,
		int* hlb,
		int* hrb,

		int* hflow){
	cout << "start run_auction\n";
	int* dedges;
	int* dcost;
	int* dcostRaw;
	int* dg;
	int* dgraw;
	int* dlb;
	int* drb;

	int* dprice;

	int* dflow;

	hipMalloc((void **)&dedges, EDGESIZE*2*sizeof(int));
	hipMalloc((void **)&dcost, SIZE*SIZE*sizeof(int));
	hipMalloc((void **)&dcostRaw, SIZE*SIZE*sizeof(int));
	hipMalloc((void **)&dg, SIZE*sizeof(int));
	hipMalloc((void **)&dgraw, SIZE*sizeof(int));
	hipMalloc((void **)&dlb, SIZE*SIZE*sizeof(int));
	hipMalloc((void **)&drb, SIZE*SIZE*sizeof(int));

	hipMalloc((void **)&dprice, SIZE*sizeof(int));

	hipMalloc((void **)&dflow, SIZE*SIZE*sizeof(int));


	hipMemcpy(dedges, hedges, EDGESIZE*2*sizeof(int), hipMemcpyHostToDevice);
	
	hipMemcpy(dcost, hcost, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dcostRaw, hcost, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(dg, hg, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dgraw, hg, SIZE*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(dlb, hlb, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(drb, hrb, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);

	hipProfilerStart();
	cout << "start kernel\n";
	auction_algorithm_kernel<<<1,threadNum>>>
		(
		numNodes,
		numEdges,
		threadNum,
		dC,
		dedges,
		dcost,
		dcostRaw,
		dg,
		dgraw,
		dlb,
		drb,
		dprice,
		dflow);
	hipProfilerStop();
	hipDeviceSynchronize();

	hipMemcpy(hflow, dflow, SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost);
	
	int ans = 0;
	for(int i = 0; i < numNodes; i++){
		for(int j = 0; j < numNodes; j++){
//			ans += hflow[i*numNodes + j]*hcost[i*numNodes+ j];
//			cout << hflow[i*numNodes + j] << " ";
		}
	}
	cout << "ans:  " << ans << endl;

}


void initmy(
		int *dc,
		int *edges,
		int *cost,
		int *hg,
		int *lb,
		int *rb){
	cout << "start read in graph..\n";
	int tnumNodes;
	int tCapacity = 0;
	int tmaxCost = 0;
	cin >> tnumNodes;
	cout << "tnumNodes: "<< tnumNodes << endl;
	memset(cost, MAXMY, sizeof(cost));
	memset(edges, 0, sizeof(edges));
	memset(hg, 0, sizeof(hg));
	char a;
	int fid;
	int aNUm;
	cin >> aNUm;
//	cout << "aNUm " << aNUm << endl;
	for(int i = 0; i < aNUm; i++){
		cin >> a >> fid;
		cin >> hg[fid-1];
//		cout << a << " " << fid << " " << g[fid-1] << endl;
	}
	int ti,tj;
	int edgeNum = 0;
	while(true){
		cin >> a >> ti >> tj;
		if(ti == tj&&ti==0){
			break;
		}
		ti--;tj--;
		edges[edgeNum*2] = ti;
		edges[edgeNum*2 + 1] = tj;
		edgeNum++;

		cin >> lb[ti*SIZE + tj] >> rb[ti*SIZE + tj] >>  cost[ti*SIZE + tj] ;
//		cout << a << "\t" << ti << " " << tj << " " << cost[ti*SIZE + tj] <<" " << lb[ti*SIZE + tj] << " " << rb[ti*SIZE + tj] <<  endl;
//		cost[ti][tj] *= nodeNum;
//		cost[ti][tj] %= 4000;
		tmaxCost = max(cost[ti*SIZE + tj], tmaxCost);
		tCapacity = max(rb[ti*SIZE + tj], tCapacity);
	}
	*dc = tmaxCost;
	cout << "read end\n";
}

int main(int argc, char *argv[]){
	int threadNum = 1024;
	int numNodes = SIZE;
	int numEdges = EDGESIZE;
	int hC;
	int *hedges = new int[EDGESIZE*2];
	int *hcost = new int[SIZE*SIZE];
	int *hg = new int[SIZE];
	int *hlb = new int[SIZE*SIZE];
	int *hrb = new int[SIZE*SIZE];

	int *hflow = new int[SIZE*SIZE];
	memset(hflow, 0, sizeof(hflow));

	initmy(
		&hC,
		hedges,
		hcost,
		hg,
		hlb,
		hrb
		
	);

	run_auction(
		numNodes,
		numEdges,
		threadNum,
		hC,

		hedges,
		hcost,
		hg,
		hlb,
		hrb,

		hflow
	);
	return 0;
}
