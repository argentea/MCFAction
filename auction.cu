#include "hip/hip_runtime.h"
#include "auction.cuh"

#define MAXMY 0x3f3f3f3f

__device__ int kflag;
#if DEBUG
__device__ int tans;
#endif
__device__ int minRise;
__device__ int kpoCount;
__device__ int knaCount;
__device__ int kpushListPo[SIZE][2];
__device__ int kpushListNa[SIZE][2];
__device__ bool knodesRisePrice[SIZE];
//pushlist is not good
__device__ void pushFlow(
		Graph &G,
		const int lnodes,
		const int rnodes,
		const int ledges,
		const int redges,
		const int epsilon,
		const int knumNodes
		){
#if FULLDEBUG
	if(threadIdx.x ==0){
		printf("in pushFlow\n");
	}
	__syncthreads();
#endif
	if(threadIdx.x ==0){
		kpoCount = 0;
		knaCount = 0;
	}
	__syncthreads();

	for(int i = ledges; i < redges; i++){
		int ti,tj,tindex;
		ti = G.edge2source(i);
		tj = G.edge2sink(i);
		if(G.atCost(ti,tj) - G.atPrice(ti) + G.atPrice(tj) + epsilon == 0&&G.atGrow(ti) >0){
			tindex = atomicAdd(&kpoCount, 1);
			kpushListPo[tindex][0] = ti;
			kpushListPo[tindex][1] = tj;
			continue;
		}
		if(G.atCost(ti,tj) - G.atPrice(ti) + G.atPrice(tj) - epsilon == 0&&G.atGrow(tj) > 0){
			tindex = atomicAdd(&knaCount, 1);
			kpushListNa[tindex][0] = tj;
			kpushListNa[tindex][1] = ti;
			continue;
		}
	}
#if FULLDEBUG
	if(threadIdx.x ==0){
		printf("get pushList\n");
	}
	__syncthreads();
#endif
	__syncthreads();
	int delta,tmpi,tmpj;
	if(threadIdx.x == 0){
		for(int i = 0; i < kpoCount; i++){
			tmpi = kpushListPo[i][0];
			tmpj = kpushListPo[i][1];
			delta = min(G.atGrow(tmpi), G.atRb(tmpi,tmpj) - G.atFlow(tmpi, tmpj));
			G.setFlow(tmpi, tmpj, G.atFlow(tmpi, tmpj) + delta);
			G.atomicSubGrow(tmpi, delta);
			G.atomicAddGrow(tmpj, delta);
		}
		for(int i = 0; i < knaCount; i++){
			tmpi = kpushListNa[i][0];
			tmpj = kpushListNa[i][1];
			delta = min(G.atGrow(tmpi), G.atFlow(tmpj, tmpi) - G.atLb(tmpj,tmpi));
			G.setFlow(tmpj, tmpi, G.atFlow(tmpj, tmpi) - delta);
			G.atomicSubGrow(tmpi, delta);
			G.atomicAddGrow(tmpj, delta);
		}
	}
	__syncthreads();
#if FULLDEBUG
		if(threadIdx.x == 0){
			printf("out pushFlow\n");
		}
		__syncthreads();
#endif

	return ;
}
__device__ void priceRise(
		Graph &G,
		const int lnodes,
		const int rnodes,
		const int ledges,
		const int redges,
		const int epsilon,
		const int knumNodes
		){
#if FULLDEBUG
		if(threadIdx.x == 0){
			printf("in priceRise\n");
		}
		__syncthreads();
#endif

	int ti,tj,tmpa,tmpb;
	for(int i = lnodes; i < rnodes; i++){
		if(G.atGrow(i) > 0){
			knodesRisePrice[i] = true;
		}else {
			knodesRisePrice[i] = false;
		}
	}
	__syncthreads();
	for(int i = ledges; i < redges; i++){
		ti = G.edge2source(i);
		tj = G.edge2sink(i);
		if(knodesRisePrice[ti]!=knodesRisePrice[tj]){
			if(G.atFlow(ti,tj) < G.atRb(ti,tj)&&knodesRisePrice[ti]){
				tmpb = G.atPrice(tj) + G.atCost(ti, tj) + epsilon - G.atPrice(ti);
				if(tmpb >= 0){
					atomicMin(&minRise, tmpb);
				}
			}
			if(G.atFlow(ti,tj) > G.atLb(ti,tj)&&knodesRisePrice[tj]){
				tmpa = G.atPrice(ti) - G.atCost(ti, tj) + epsilon - G.atPrice(tj);
				if(tmpa >= 0){
					atomicMin(&minRise, tmpa);
				}
			}
		}
	}
#if FULLDEBUG
		if(threadIdx.x == 0){
			printf("out priceRise\n");
		}
		__syncthreads();
#endif
	__syncthreads();

}
__global__ void __launch_bounds__(1024, 1)
auction_algorithm_kernel(
		Graph G,
		const int kthreadNum
){
	const int threadId = threadIdx.x;

	
	if(threadId == 0){
		printf("in kernel\n");
	}
	__syncthreads();

	int knumNodes = G.getNodesNum();
	int knumEdges = G.getEdgesNum();

	int kepsilon = 1;
	int edgesDivThread;
	int edgesModThread;
	//[edgesl,edgesr) is the range of edges that the thread produre
	int ledges;
	int redges;

	int nodesDivThread;
	int nodesModThread;
	int lnodes;
	int rnodes;

	int totalIteratorNum = 0;
	int iteratorNum = 0;
	int scalingFactor = 2;
	int costScale = 9;
	int gdelta = 0;

	int kti;
	int ktj;

	edgesDivThread = knumEdges/kthreadNum;
	edgesModThread = knumEdges%kthreadNum;
	
	if(threadId < edgesModThread){
		ledges = threadId*(edgesDivThread + 1);
		redges = (threadId + 1)*(edgesDivThread + 1);
	}else {
		ledges = threadId*edgesDivThread + edgesModThread;
		redges = (threadId + 1)*edgesDivThread + edgesModThread;
	}
	
	nodesDivThread = knumNodes/kthreadNum;
	nodesModThread = knumNodes%kthreadNum;

	if(threadId < nodesModThread){
		lnodes = threadId*(nodesDivThread + 1);
		rnodes = (threadId + 1)*(nodesDivThread + 1);
	}else{
		lnodes = threadId*nodesDivThread + nodesModThread;
		rnodes = (threadId + 1)*nodesDivThread + nodesModThread;
	}

	while(costScale >= 0){
#if DEBUG
		if(threadId == 0){
			printf("cost scale: %d\n",costScale);
		}
#endif
		for(int i = lnodes; i < rnodes; i++){
			G.setGrow(i , G.atGrowRaw(i));
		}

		int ktmp = 1<<costScale;

		for(int i = ledges; i < redges; i++){
			kti = G.edge2source(i);
			ktj = G.edge2sink(i);
			G.setFlow(kti, ktj, 0);
			if(G.atCostRaw(kti,ktj) <= G.getMaxCost()){
				G.setCost(kti, ktj, G.atCostRaw(kti,ktj)/ktmp);
			}
		}
		for(int i = lnodes; i < rnodes; i++){
			G.setPrice(i, G.atPrice(i)*(1 << gdelta));
		}
		__syncthreads();
		for(int i = ledges; i < redges; i++){
			kti = G.edge2source(i);
			ktj = G.edge2sink(i);
			if(G.atCost(kti, ktj) - G.atPrice(kti) + G.atPrice(ktj) + kepsilon <= 0){
				G.atomicSubGrow(kti, G.atRb(kti,ktj));
				G.atomicAddGrow(ktj, G.atRb(kti,ktj));
				G.setFlow(kti, ktj, G.atRb(kti,ktj));
			}
		}
		iteratorNum = 0;
		if(threadId == 0)
		{
			kflag = true;
		}
		__syncthreads();

		for(int i = lnodes; i < rnodes; i++){
			if(G.atGrow(i) != 0){
				atomicAnd(&kflag, 0);
			}
		}
		__syncthreads();

		while(!kflag){
#if FULLDEBUG
			if(threadId == 0){
				printf("iteration : %d\n", iteratorNum);
			}
			__syncthreads();
#endif
			pushFlow(
					G,
				lnodes,
				rnodes,
				ledges,
				redges,
				kepsilon,
				knumNodes
				);
			if(threadId == 0){
				minRise = MAXMY;
			}
			__syncthreads();
			priceRise(
					G,
				lnodes,
				rnodes,
				ledges,
				redges,
				kepsilon,
				knumNodes
				);
			__syncthreads();
#if FULLDEBUG
			if(threadId == 0){
				printf("minRise: %d\n", minRise);
			}
			__syncthreads();
#endif
			if(threadId == 0){
				if(minRise == MAXMY){
					minRise = 0;
				}
			}

			__syncthreads();
			for(int i = lnodes; i < rnodes; i++){
				if(knodesRisePrice[i]){
					G.setPrice(i, G.atPrice(i) + minRise);
				}
			}
			__syncthreads();
			iteratorNum++;
			totalIteratorNum++;
			if(threadId == 0)
			{
				kflag = true;
			}
			for(int i = lnodes; i < rnodes; i++){
				if(G.atGrow(i) != 0){
					atomicAnd(&kflag, 0);
				}
			}
			__syncthreads();

		}

#if DEBUG
		if(threadId == 0){
			tans = 0;
		}
		__syncthreads();
		for(int i = ledges; i < redges; i++){
			kti = G.edge2source(i);
			ktj = G.edge2sink(i);
			atomicAdd(&tans, G.atFlow(kti,ktj)*G.atCostRaw(kti,ktj));
		}
		if(threadId == 0){
			printf("inner loop out\n");
			printf("temporary ans: %d\n",tans);
			printf("cost scale: %d\n", costScale);
			printf("iteratorNum: %d\n", iteratorNum);
		}
		__syncthreads();
#endif
		if(costScale ==0){
			break;
		}
		gdelta = costScale - max(0, costScale - scalingFactor);
		costScale = max(0, costScale - scalingFactor);
	}


	if(threadId == 0)
	{
		printf("totalIteratorNum: %d\n", totalIteratorNum);
		printf("kenerl end\n");
	}
	__syncthreads();
}
hr_clock_rep timer_start, timer_mem, timer_stop;

void run_auction(
		Graph auctionGraph,
		int threadNum,
		int* hflow){
	std::cout << "start run_auction\n";
	timer_start = get_globaltime();

	timer_mem = get_globaltime();
	hipProfilerStart();
	std::cout << "start kernel\n";
	auction_algorithm_kernel<<<1,threadNum>>>
		(
		auctionGraph,
		threadNum
		);
	hipProfilerStop();
	hipDeviceSynchronize();
	timer_stop = get_globaltime();
}

int main(int argc, char *argv[]){
	int threadNum = 1024;
	int *hflow = new int[SIZE*SIZE];
	memset(hflow, 0, sizeof(hflow));

//	initmy(&hC,hedges,hcost,hg,hlb,hrb	);
	Graph auctionGraph = Graph(Graph::fakeEdgeList, "../data/data1.min");

//	Graph auctionGraph = Graph(Graph::matrix,numNodes, numEdges, hC, hedges, hcost, hlb, hrb, hg);

	run_auction(
		auctionGraph,
		threadNum,
		hflow
	);

	std::cerr << "run_acution takes "<< (timer_stop - timer_start)*get_timer_period() << "ms totally.\n";
	std::cerr << "memory copy takes "<< (timer_mem - timer_start)*get_timer_period() << "ms totally.\n";
	std::cerr << "kernel takes "<< (timer_stop - timer_mem)*get_timer_period() << "ms totally.\n";
	return 0;
}
