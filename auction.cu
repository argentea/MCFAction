#include "hip/hip_runtime.h"
#include "auction.cuh"

#define MAXMY 0x3f3f3f3f

struct PushEdge
{
	int edge;
	int delta;
	bool direct;
};

struct AuctionState
{
	struct PushEdge* kpushList;

	int* kpushListDelta;
	int* kpushListFlag;
    bool* knodesRisePrice; ///< length of #nodes 

    void initialize(Graph const& G)
    {
        printf("initialize state with %d nodes\n", G.getNodesNum());
		kpushList = nullptr;
        knodesRisePrice = nullptr; 

		hipError_t status = hipMalloc((void **)&kpushList, G.getEdgesNum()*sizeof(PushEdge));
        if (status != hipSuccess) 
        { 
            printf("hipMalloc failed for kpushList\n"); 
        } 

		status = hipMalloc((void **)&kpushListFlag, G.getNodesNum()*sizeof(int));
        if (status != hipSuccess) 
        { 
            printf("hipMalloc failed for kpushListFlag\n"); 
        } 
		status = hipMalloc((void **)&kpushListDelta, G.getEdgesNum()*sizeof(bool));
		if (status != hipSuccess)
		{
			printf("hipMalloc failed for kpushListDelta\n");
		}
        status = hipMalloc((void **)&knodesRisePrice, G.getNodesNum()*sizeof(bool));
        if (status != hipSuccess) 
        { 
            printf("hipMalloc failed for knodesRisePrice\n"); 
        } 
    }

    void destroy()
    {
        hipFree(knodesRisePrice);
		hipFree(kpushListDelta);
		hipFree(kpushList);
    }
};

//pushlist is not good
__device__ void pushFlow(
		Graph &G,
        AuctionState& state, 
		const int lnodes,
		const int rnodes,
        const int node_step, 
		const int ledges,
		const int redges,
        const int edge_step, 
		const int epsilon,
		const int knumNodes, 
        int& kpoCount, 
        int& knaCount,
		int& kpushCount
		){
#if FULLDEBUG
	if(threadIdx.x ==0){
		printf("in pushFlow\n");
	}
	__syncthreads();
#endif
	if(threadIdx.x ==0){
		kpoCount = 0;
		knaCount = 0;
		kpushCount = 0;
	}
	__syncthreads();

	for(int i = ledges; i < redges; i += edge_step){
		int ti,tj,tindex,mindex;
        auto const& edge = G.edge(i); 
		ti = edge.source;
		tj = edge.sink;
        int value = G.atCost(i) - G.atPrice(ti) + G.atPrice(tj);
		if(value + epsilon == 0 && G.atGrow(ti) >0){
			tindex = atomicAdd(&kpoCount, 1);
			
			mindex = atomicAdd(&kpushCount, 1);


			state.kpushList[mindex].edge = i;
			state.kpushList[mindex].delta = G.atRb(i) - G.atFlow(i);
			state.kpushList[mindex].direct = true;
			state.kpushListDelta[mindex] = G.atRb(i) - G.atFlow(i);
		}
		else if (value - epsilon == 0 && G.atGrow(tj) > 0){
			tindex = atomicAdd(&knaCount, 1);

			mindex = atomicAdd(&kpushCount, 1);
			state.kpushList[mindex].edge = i;
			state.kpushList[mindex].direct = false;
			state.kpushList[mindex].delta = G.atFlow(i) - G.atLb(i);

			state.kpushListDelta[mindex] = G.atLb(i) - G.atFlow(i);
		}
	}
#if FULLDEBUG
	if(threadIdx.x ==0){
		printf("get pushList\n");
	}
	__syncthreads();
#endif
	__syncthreads();
	int delta,tmpi,tmpj,tmpk;

	int tdivid = kpushCount / blockDim.x;
	int tmod = kpushCount % blockDim.x;
	int tlb,trb,told;
	if(threadIdx.x < tmod){
		tlb = threadIdx.x * (tdivid + 1);
		trb = (threadIdx.x + 1) * (tdivid + 1);
	}else{
		tlb = threadIdx.x * tdivid + tmod;
		trb = (threadIdx.x + 1)*tdivid + tmod;
	}
/*	while(todoCheck){
		for(int i = tlb; i < trb; i++)
		{

		}	
	}
*/
	if(threadIdx.x == 0){
		for(int i = 0; i < kpushCount; i++){
			tmpk = state.kpushList[i].edge;
			auto const& edge = G.edge(tmpk);
			tmpi = edge.source;
			tmpj = edge.sink;
			if(state.kpushList[i].direct){
				delta = min(G.atGrow(tmpi), G.atRb(tmpk) - G.atFlow(tmpk));
			}else{
				delta = -min(G.atGrow(tmpj), G.atFlow(tmpk) - G.atLb(tmpk));
			}
			G.setFlow(tmpk, G.atFlow(tmpk) + delta);
			G.atomicSubGrow(tmpi, delta);
			G.atomicAddGrow(tmpj, delta);
		}
/*
		for(int i = 0; i < kpoCount; i++){
            tmpk = state.kpushListPo[i]; 
            auto const& edge = G.edge(tmpk); 
            tmpi = edge.source; 
            tmpj = edge.sink; 
			delta = min(G.atGrow(tmpi), G.atRb(tmpk) - G.atFlow(tmpk));
			G.setFlow(tmpk, G.atFlow(tmpk) + delta);
			G.atomicSubGrow(tmpi, delta);
			G.atomicAddGrow(tmpj, delta);
		}
		for(int i = 0; i < knaCount; i++){
            tmpk = state.kpushListNa[i]; 
            auto const& edge = G.edge(tmpk); 
            tmpi = edge.sink; 
            tmpj = edge.source; 
			G.setFlow(tmpk, G.atFlow(tmpk) - delta);
			G.atomicSubGrow(tmpi, delta);
			G.atomicAddGrow(tmpj, delta);
		}*/
	}
	__syncthreads();
#if FULLDEBUG
		if(threadIdx.x == 0){
			printf("out pushFlow\n");
		}
		__syncthreads();
#endif

	return ;
}
__device__ void priceRise(
		Graph &G,
        AuctionState& state, 
		const int lnodes,
		const int rnodes,
        const int node_step, 
		const int ledges,
		const int redges,
        const int edge_step, 
		const int epsilon,
		const int knumNodes, 
        int& minRise
		){
#if FULLDEBUG
		if(threadIdx.x == 0){
			printf("in priceRise\n");
		}
		__syncthreads();
#endif

	int ti,tj,tmpa,tmpb;
	for(int i = lnodes; i < rnodes; i += node_step){
		if(G.atGrow(i) > 0){
			state.knodesRisePrice[i] = true;
		}else {
			state.knodesRisePrice[i] = false;
		}
	}
	__syncthreads();
	for(int i = ledges; i < redges; i += edge_step){
        auto const& edge = G.edge(i);
		ti = edge.source;
		tj = edge.sink;
		if(state.knodesRisePrice[ti] != state.knodesRisePrice[tj]){
			if(G.atFlow(i) < G.atRb(i) && state.knodesRisePrice[ti]){
				tmpb = G.atPrice(tj) + G.atCost(i) + epsilon - G.atPrice(ti);
				if(tmpb >= 0){
					atomicMin(&minRise, tmpb);
				}
			}
			if(G.atFlow(i) > G.atLb(i) && state.knodesRisePrice[tj]){
				tmpa = G.atPrice(ti) - G.atCost(i) + epsilon - G.atPrice(tj);
				if(tmpa >= 0){
					atomicMin(&minRise, tmpa);
				}
			}
		}
	}
#if FULLDEBUG
		if(threadIdx.x == 0){
			printf("out priceRise\n");
		}
		__syncthreads();
#endif
	__syncthreads();

}
__global__ void __launch_bounds__(1024)
auction_algorithm_kernel(
		Graph G, 
        AuctionState state 
){
	__shared__ int kepsilon;
	__shared__ int totalIteratorNum;
	__shared__ int iteratorNum;
	__shared__ int scalingFactor;
	__shared__ int costScale;
	__shared__ int gdelta;
	__shared__ int knumNodes;
	__shared__ int knumEdges;
	__shared__ int edgesDivThread;
	__shared__ int nodesDivThread;
    __shared__ int kflag; 
    __shared__ int minRise;
    __shared__ int kpoCount;
    __shared__ int knaCount;
	__shared__ int kpushCount;
	__shared__ int kpushFlag;
#if DEBUG
    __shared__ int tans;
#endif

	const int threadId = threadIdx.x;
    if (threadId == 0) {
        kepsilon = 1; 
        totalIteratorNum = 0; 
        iteratorNum = 0; 
        scalingFactor = 2; 
        costScale = 9; 
        gdelta = 0; 
        knumNodes = G.getNodesNum();
        knumEdges = G.getEdgesNum();
        edgesDivThread = max(knumEdges / blockDim.x, 1);
        nodesDivThread = max(knumNodes / blockDim.x, 1);

		printf("in kernel\n");
    }
    __syncthreads();

	//[edgesl,edgesr) is the range of edges that the thread produre
	const int ledges = threadId * edgesDivThread;
	const int redges = min(ledges + edgesDivThread, knumEdges);
    const int edge_step = 1; 

	const int lnodes = threadId * nodesDivThread;
	const int rnodes = min(lnodes + nodesDivThread, knumNodes);
    const int node_step = 1; 

	int kti;
	int ktj;

	while(costScale >= 0){
#if DEBUG
		if(threadId == 0){
			printf("cost scale: %d\n",costScale);
		}
#endif
		for(int i = lnodes; i < rnodes; i += node_step){
			G.setGrow(i , G.atGrowRaw(i));
		}

		int ktmp = 1<<costScale;

		for(int i = ledges; i < redges; i += edge_step){
			G.setFlow(i, 0);
			if(G.atCostRaw(i) <= G.getMaxCost()){
				G.setCost(i, G.atCostRaw(i)/ktmp);
			}
		}
		for(int i = lnodes; i < rnodes; i++){
			G.setPrice(i, G.atPrice(i)*(1 << gdelta));
		}
		__syncthreads();
		for(int i = ledges; i < redges; i += edge_step){
            auto const& edge = G.edge(i);
			kti = edge.source;
			ktj = edge.sink;
			if(G.atCost(i) - G.atPrice(kti) + G.atPrice(ktj) + kepsilon <= 0){
				G.atomicSubGrow(kti, G.atRb(i));
				G.atomicAddGrow(ktj, G.atRb(i));
				G.setFlow(i, G.atRb(i));
			}
		}
		if(threadId == 0)
		{
            iteratorNum = 0;
			kflag = true;
		}
		__syncthreads();

		for(int i = lnodes; i < rnodes; i += node_step){
			if(G.atGrow(i) != 0){
				atomicAnd(&kflag, 0);
			}
		}
		__syncthreads();

		while(!kflag){
#if FULLDEBUG
			if(threadId == 0){
				printf("iteration : %d\n", iteratorNum);
			}
			__syncthreads();
#endif
            pushFlow(
                    G,
                    state, 
                    lnodes,
                    rnodes,
                    node_step, 
                    ledges,
                    redges,
                    edge_step, 
                    kepsilon,
                    knumNodes, 
                    kpoCount, 
                    knaCount,
					kpushCount
                    );
			if(threadId == 0){
				minRise = MAXMY;
			}
			__syncthreads();
            priceRise(
                    G,
                    state, 
                    lnodes,
                    rnodes,
                    node_step, 
                    ledges,
                    redges,
                    edge_step, 
                    kepsilon,
                    knumNodes, 
                    minRise
                    );
			__syncthreads();
#if DEBUG
			if(threadId == 0){
				if(minRise == 0)
				printf("iteration : %d  minRise: %d\n", iteratorNum ,minRise);
			}
			__syncthreads();
#endif
			if(threadId == 0){
				if(minRise == MAXMY){
					minRise = 0;
				}
			}

			__syncthreads();
			for(int i = lnodes; i < rnodes; i += node_step){
				if(state.knodesRisePrice[i]){
					G.setPrice(i, G.atPrice(i) + minRise);
				}
			}
			__syncthreads();
			if(threadId == 0)
			{
                iteratorNum++;
                totalIteratorNum++;
				kflag = true;
			}
			for(int i = lnodes; i < rnodes; i += node_step){
				if(G.atGrow(i) != 0){
					atomicAnd(&kflag, 0);
				}
			}
			__syncthreads();

		}

#if DEBUG
		if(threadId == 0){
			tans = 0;
		}
		__syncthreads();
		for(int i = ledges; i < redges; i += edge_step){
			atomicAdd(&tans, G.atFlow(i)*G.atCostRaw(i));
		}
		if(threadId == 0){
			printf("inner loop out\n");
			printf("temporary ans: %d\n",tans);
			printf("cost scale: %d\n", costScale);
			printf("iteratorNum: %d\n", iteratorNum);
		}
		__syncthreads();
#endif
		if(costScale ==0){
			break;
		}
        if (threadId == 0) {
            gdelta = costScale - max(0, costScale - scalingFactor);
            costScale = max(0, costScale - scalingFactor);
        }
        __syncthreads();
	}

	if(threadId == 0)
	{
		printf("totalIteratorNum: %d\n", totalIteratorNum);
		printf("kenerl end\n");
	}
}

hr_clock_rep timer_start, timer_mem, timer_stop;
void run_auction(
		Graph auctionGraph,
		int threadNum,
		int* hflow){
	std::cout << "start run_auction\n";

	hipProfilerStart();
	std::cout << "start kernel\n";
    AuctionState state; 
    state.initialize(auctionGraph);
	auction_algorithm_kernel<<<1,threadNum>>>
		(
		auctionGraph, 
        state
		);
    state.destroy();
	hipProfilerStop();
	hipDeviceSynchronize();
	timer_stop = get_globaltime();
}

int main(int argc, char *argv[]){
	int threadNum = 1024;
//	initmy(&hC,hedges,hcost,hg,hlb,hrb	);
	timer_start = get_globaltime();
	Graph auctionGraph = Graph(Graph::edgeList, argv[1]);
	timer_mem = get_globaltime();

//	Graph auctionGraph = Graph(Graph::matrix,numNodes, numEdges, hC, hedges, hcost, hlb, hrb, hg);

    std::vector<int> hflow (auctionGraph.getNodesNum() * auctionGraph.getNodesNum(), 0);
	run_auction(
		auctionGraph,
		threadNum,
		hflow.data()
	);

	std::cerr << "run_acution takes "<< (timer_stop - timer_start)*get_timer_period() << "ms totally.\n";
	std::cerr << "memory copy takes "<< (timer_mem - timer_start)*get_timer_period() << "ms totally.\n";
	std::cerr << "kernel takes "<< (timer_stop - timer_mem)*get_timer_period() << "ms totally.\n";
	return 0;
}
