#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <string>
#include <fstream>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <chrono>
#define MAXMY 0x3f3f
#define SIZE 256
#define EDGESIZE 2048
#define DEBUG 1
using namespace std;

__device__ unsigned int justForTest = 0;
__device__ void dcostScalingInit(
		const int costScale,
		const int gdelta,
		const int ledge,
		const int redge,
		const int lnode,
		const int rnode,
		const int* edges,
		const int* costRaw,
		int* cost,
		int* price){
	int ti,tj;
	for(int i = ledge; i < redge; i++){
		ti = edges[i*2 + 0];
		tj = edges[i*2 + j];
		if(costRaw 
}
__global__ void __launch_bounds__(1024, 1)
auction_algorithm_kernel(
		const int knumNodes,
		const int knumEdges,
		const int kthreadNum,

		const int* kedges,
		int* kcost,
		const int* kcostRaw,
		int* kg,
		const int* kgraw,
		const int* klb,
		const int* krb,
		const int* kprice,
		int* kflow){
	const int threadId = threadIdx.x;
#if DEBUG
//	if(threadId == 0){
//		printf("in kernel\n");
//	}
//	printf("threadId: %d\n", threadId);
#endif
	int edgesDivThread;
	int edgesModThread;
	//[edgesl,edgesr) is the range of edges that the thread produre
	int ledges;
	int redges;

	int nodesDivThread;
	int nodesModThread;
	int lnodes;
	int rnodes;

	int totalIteratorNum = 0;
	int iteratorNum = 0;
	int allIterater = 0;
	int tmpa = 0;
	int tmpb = 0;
	int tmpi = 0;
	int scalingFactor = 2;
	int costScale = 9;
	int gdelta = 0;

	int kti;
	int ktj;

	edgesDivThread = knumEdges/kthreadNum;
	edgesModThread = knumEdges%kthreadNum;
	
	if(threadId < edgesModThread){
		ledges = threadId*(edgesDivThread + 1);
		redges = (threadId + 1)*(edgesDivThread + 1);
	}else {
		ledges = threadId*edgesDivThread + edgesModThread;
		redges = (threadId + 1)*edgesDivThread + edgesModThread;
	}
	
	nodesDivThread = knumNodes/kthreadNum;
	nodesModThread = knumNodes%kthreadNum;

	if(threadId < nodesModThread){
		lnodes = threadId*(nodesDivThread + 1);
		rnodes = (threadId + 1)*(nodesDivThread + 1);
	}else{
		lnodes = threadId*nodesDivThread + nodesModThread;
		rnodes = (threadId + 1)*nodesDivThread + nodesModThread;
	}
#if DEBUG
//	printf("threadId: %d, ledges: %d, redges: %d\n", threadId, ledges, redges);
//	__syncthreads();
//	for(int i = ledges; i < redges; i++){
//		kflow[kedges[i*2 + 0] * knumNodes + kedges[i*2 + 1]] = atomicAdd(&justForTest, 1);
//		printf("%d\n", kflow[kedges[i*2 + 0] * knumNodes + kedges[i*2 + 1]]);
//	}
//	__syncthreads();
#endif
	
	while(costScale >= 0){
		int ktmp = 1<<costScale;

		for(int i = ledges; i < redges; i++){
			kti = kedges[i*2 + 0];
			ktj = kedges[i*2 + 1];
			kflow[kti * knumNodes + ktj] = 0;
			if(kcostRaw[kti*knumNodes + ktj] <= C){
				kcost[kti*knumNodes + ktj] = kcostRaw[kti*knumNodes + ktj]/ktmp;
			}
		}
		for(int i = lnodes; i < rnodes; i++){
			kprice[i]*=(1 << gdelta);
		}



	}

		


	if(threadId == 0)
	{
		printf("kenerl end\n");
	}
}

void run_auction(
		int numNodes,
		int numEdges,
		int threadNum,

		int* hedges,
		int* hcost,
		int* hg,
		int* hlb,
		int* hrb,

		int* hflow){
	cout << "start run_auction\n";
	int* dedges;
	int* dcost;
	int* dcostRaw;
	int* dg;
	int* dgraw;
	int* dlb;
	int* drb;

	int* dprice;

	int* dflow;

	hipMalloc((void **)&dedges, EDGESIZE*2*sizeof(int));
	hipMalloc((void **)&dcost, SIZE*SIZE*sizeof(int));
	hipMalloc((void **)&dcostRaw, SIZE*SIZE*sizeof(int));
	hipMalloc((void **)&dg, SIZE*sizeof(int));
	hipMalloc((void **)&dgraw, SIZE*sizeof(int));
	hipMalloc((void **)&dlb, SIZE*SIZE*sizeof(int));
	hipMalloc((void **)&drb, SIZE*SIZE*sizeof(int));

	hipMalloc((void **)&dprice, SIZE*sizeof(int));

	hipMalloc((void **)&dflow, SIZE*SIZE*sizeof(int));


	hipMemcpy(dedges, hedges, EDGESIZE*2*sizeof(int), hipMemcpyHostToDevice);
	
	hipMemcpy(dcost, hcost, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dcostRaw, hcost, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(dg, hg, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dgraw, hg, SIZE*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(dlb, hlb, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(drb, hrb, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);

	hipProfilerStart();
	cout << "start kernel\n";
	auction_algorithm_kernel<<<1,threadNum>>>
		(
		numNodes,
		numEdges,
		threadNum,
		dedges,
		dcost,
		dcostRaw,
		dg,
		dgraw,
		dlb,
		drb,
		dprice,
		dflow);
	hipProfilerStop();
	hipDeviceSynchronize();

	hipMemcpy(hflow, dflow, SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost);
	
	int ans = 0;
	for(int i = 0; i < numNodes; i++){
		for(int j = 0; j < numNodes; j++){
//			ans += hflow[i*numNodes + j]*hcost[i*numNodes+ j];
//			cout << hflow[i*numNodes + j] << " ";
		}
	}
	cout << "ans:  " << ans << endl;

}


void initmy(
		int *edges,
		int *cost,
		int *hg,
		int *lb,
		int *rb){
	cout << "start read in graph..\n";
	int tnumNodes;
	int tCapacity = 0;
	int tmaxCost = 0;
	cin >> tnumNodes;
	memset(cost, MAXMY, sizeof(cost));
	memset(edges, 0, sizeof(edges));
	memset(hg, 0, sizeof(hg));
	char a;
	int fid;
	int aNUm;
	cin >> aNUm;
//	cout << "aNUm " << aNUm << endl;
	for(int i = 0; i < aNUm; i++){
		cin >> a >> fid;
		cin >> hg[fid-1];
//		cout << a << " " << fid << " " << g[fid-1] << endl;
	}
	int ti,tj;
	int edgeNum = 0;
	while(true){
		cin >> a >> ti >> tj;
		if(ti == tj&&ti==0){
			break;
		}
		ti--;tj--;
		edges[edgeNum*2] = ti;
		edges[edgeNum*2 + 1] = tj;
		edgeNum++;

		cin >> lb[ti*SIZE + tj] >> rb[ti*SIZE + tj] >>  cost[ti*SIZE + tj] ;
//		cout << a << "\t" << ti << " " << tj << " " << cost[ti*SIZE + tj] <<" " << lb[ti*SIZE + tj] << " " << rb[ti*SIZE + tj] <<  endl;
//		cost[ti][tj] *= nodeNum;
//		cost[ti][tj] %= 4000;
		tmaxCost = max(cost[ti*SIZE + tj], tmaxCost);
		tCapacity = max(rb[ti*SIZE + tj], tCapacity);
	}
	cout << "read end\n";
}

int main(int argc, char *argv[]){
	int threadNum = 1024;
	int numNodes = SIZE;
	int numEdges = EDGESIZE;
	int *hedges = new int[EDGESIZE*2];
	int *hcost = new int[SIZE*SIZE];
	int *hg = new int[SIZE];
	int *hlb = new int[SIZE*SIZE];
	int *hrb = new int[SIZE*SIZE];

	int *hflow = new int[SIZE*SIZE];
	memset(hflow, 0, sizeof(hflow));

	initmy(
		hedges,
		hcost,
		hg,
		hlb,
		hrb
	);

	run_auction(
		numNodes,
		numEdges,
		threadNum,

		hedges,
		hcost,
		hg,
		hlb,
		hrb,

		hflow
	);
	return 0;
}
