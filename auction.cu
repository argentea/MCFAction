#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <string>
#include <fstream>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <chrono>
#define MAXMY 0x3f3f
#define SIZE 256
#define EDGESIZE 2048

__global__ void __launchbounds__(1024, 4)
auction_algorithm_kernel(
		const int knumNodes,
		const int knumEdges,

		const int* kedges,
		const int* kcost,
		const int* kcostRaw,
		const int* kg,
		const int* kgraw,
		const int* klb,
		const int* krb,
		const int* kprice,
		const int* kflow){
	const int threadId = threadIdx.x;
	int totalIteratorNum = 0;
	int iteratorNum = 0;
	int allIterater = 0;
	int tmpa = 0;
	int tmpb = 0;
	int tmpi = 0;
	scalingFactor = 2;
	int costScale = 9;
	while(costScale >= 0){
		
	}
}

void run_auction(
		int numNodes,
		int numEdges,

		int* hedges,
		int* hcost,
		int* hg,
		int* hlb,
		int* hrb,

		int* hflow){
	int* dedges;
	int* dcost;
	int* dcostRaw;
	int* dg;
	int* dgraw;
	int* dlb;
	int* drb;

	int* dprice;

	int* dflow;

	hipMalloc((void **)&dedges, EDGESIZE*2*sizeof(int));
	hipMalloc((void **)&dcost, SIZE*SIZE*sizeof(int));
	hipMalloc((void **)&dcostRaw, SIZE*SIZE*sizeof(int));
	hipMalloc((void **)&dg, SIZE*sizeof(int));
	hipMalloc((void **)&dgraw, SIZE*sizeof(int));
	hipMalloc((void **)&dlb, SIZE*SIZE*sizeof(int));
	hipMalloc((void **)&drb, SIZE*SIZE*sizeof(int));

	hipMalloc((void **)&dprice, SIZE*sizeof(int));

	hipMalloc((void **)&dflow, SIZE*SIZE*sizeof(int));


	hipMemcpy(dedges, hedges, EDGESIZE*2*sizeof(int), hipMemcpyHostToDevice);
	
	hipMemcpy(dcost, hcost, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dcostRaw, hcost, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(dg, hg, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dgraw, hg, SIZE*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(dlb, hlb, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(drb, hrb, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);

	hipProfilerStart();
	auction_algorithm_kernel<<<1,EDGESIZE>>>
		(
		numNodes,
		numEdges,
		dedges,
		dcost,
		dcostRaw,
		dg,
		dgraw,
		dlb,
		drb,
		dprice,
		dflow);
	hipProfilerStop();
	hipDeviceSynchronize();

	hipMemcpy(hflow, dflow, SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost);
	
	int ans = 0;
	for(int i = 0; i < nodeNum; i++){
		for(int j = 0; j < nodeNum; j++){
			ans += hflow[i*nodeNum + j]*hcost[i*nodeNum + j];
		}
	}
	cout << "ans:  " << ans << endl;

}


void initmy(
		int *edges,
		int *cost,
		int *hg,
		int *lb,
		int *rb){
	int tnumNodes;
	int tCapacity = 0;
	int tmaxCost = 0;
	cin >> tnumNodes;
	memset(cost, MAXMY, sizeof(cost));
	memset(edges, 0, sizeof(edges));
	memset(hg, 0, sizeof(hg));
	char a;
	int fid;
	int aNUm;
	cin >> aNUm;
//	cout << "aNUm " << aNUm << endl;
	for(int i = 0; i < aNUm; i++){
		cin >> a >> fid;
		cin >> g[fid-1];
//		cout << a << " " << fid << " " << g[fid-1] << endl;
	}
	int ti,tj;
	while(true){
		cin >> a >> ti >> tj;
		if(ti == tj&&ti==0){
			break;
		}
		ti--;tj--;
		edges[edgeNum*2] = ti;
		edges[edgeNum*2 + 1] = tj;
		edgeNum++;

		cin >> lb[ti*SIZE + tj] >> rb[ti*SIZE + tj] >>  cost[ti*SIZE + tj] ;
//		cout << a << "\t" << ti << " " << tj << " " << cost[ti][tj] <<" " << lb[ti][tj] << " " << rb[ti][tj] <<  endl;
//		cost[ti][tj] *= nodeNum;
//		cost[ti][tj] %= 4000;
		tmaxCost = max(cost[ti*SIZE + tj], tmaxCost);
		tCapacity = max(rb[ti*SIZE + tj], tCapacity);
	}

int main(int argc, char *argv[]){
	int numNodes = SIZE;
	int numEdges = EDGESIZE;
	int *hedges = new int[EDGESIZE*2];
	int *hcost = new int[SIZE*SIZE];
	int *hg = new int[SIZE];
	int *hlb = new int[SIZE*SIZE];
	int *hrb = new int[SIZE*SIZE];

	int *hflow = new int[SIZE*SIZE];
	memset(hflow, 0, sizeof(hflow));

	initmy(
		*hedges,
		*hcost,
		*hg,
		*hlb,
		*hrb
	);

	run_auction(
		numNodes,
		numEdges,

		hedges,
		hcost,
		hg,
		hlb,
		hrb,

		hflow
	);
	return 0;
}
