
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <string>
#include <fstream>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <chrono>
#define MAXMY 0x3f3f3f3f
#define SIZE 256
#define EDGESIZE 2048
#define DEBUG 1
#define FULLDEBUG 0
using namespace std;
typedef std::chrono::high_resolution_clock::rep hr_clock_rep;

inline hr_clock_rep get_globaltime(void) 
{
	using namespace std::chrono;
	return high_resolution_clock::now().time_since_epoch().count();
}

// Returns the period in miliseconds
inline double get_timer_period(void) 
{
	using namespace std::chrono;
	return 1000.0 * high_resolution_clock::period::num / high_resolution_clock::period::den;
}
__device__ int kflag;
#if DEBUG
__device__ int tans;
#endif
__device__ int minRise;
__device__ int kpoCount;
__device__ int knaCount;
__device__ int kpushListPo[SIZE][3];
__device__ int kpushListNa[SIZE][3];
__device__ bool knodesRisePrice[SIZE];
__device__ void kcheck(
		const int* kg,
		int lnodes,
		int rnodes
		){
	for(int i = lnodes; i < rnodes; i++){
		if(kg[i] != 0){
			atomicAnd(&kflag, 0);
		}
	}
	__syncthreads();
}
__device__ void printNodes(const int* nodes, int numNodes, const char* name){
	printf("*******************\n");
	printf(name);
	printf("\n");
	for(int i = 0; i < numNodes; i++){
		printf("%d\t", nodes[i]);
	}
	printf("\n*******************\n");
}
__device__ void printGraph(const int* graph, int numNodes, const char* name){
	printf("*******************\n");
	printf("%d", numNodes);
	printf(name);
//	for(int i = 0; i < numNodes; i++){
//		for(int j = 0; j < numNodes; j++){
//			printf("%d\t", graph[i*numNodes + j]);
//		}
//		printf("\n");
//	}
	printf("*******************\n");
}

__device__ unsigned int justForTest = 0;
__device__ void dcostScalingInit(
		const int costScale,
		const int gdelta,
		const int C,
		const int ledges,
		const int redges,
		const int lnodes,
		const int rnodes,
		const int knumNodes,
		const int* edges,
		const int* costRaw,
		int* cost,
		int* price){
	int ti,tj;
	for(int i = ledges; i < redges; i++){
		ti = edges[i*2 + 0];
		tj = edges[i*2 + 1];
		if(costRaw[ti*knumNodes + tj] <= C){
			cost[ti*knumNodes + tj] = costRaw[ti*knumNodes + tj]/(1 << costScale);
		}
	}
	for(int i = lnodes; i < rnodes; i++){
		price[i]*=(1 << gdelta);
	}
	return;
}
//pushlist is not good
__device__ void pushFlow(
		const int lnodes,
		const int rnodes,
		const int ledges,
		const int redges,
		const int epsilon,
		const int knumNodes,
		int* kflow,
		const int* krb,
		const int* klb,
		const int* kprice,
		const int* kcost,
		const int* kedges,
		int* kg
		){
#if FULLDEBUG
	if(threadIdx.x ==0){
		printf("in pushFlow\n");
	}
	__syncthreads();
#endif
	if(threadIdx.x ==0){
		kpoCount = 0;
		knaCount = 0;
	}
#if FULLDEBUG
	if(threadIdx.x ==0){
		printf("init pushList\n");
	}
	__syncthreads();
#endif
	for(int i = ledges; i < redges; i++){
		int ti,tj,tindex;
		ti = kedges[i*2 + 0];
		tj = kedges[i*2 + 1];
		if(kcost[i] - kprice[ti] + kprice[tj] + epsilon == 0&&kg[ti] >0){
			tindex = atomicAdd(&kpoCount, 1);
			kpushListPo[tindex][0] = ti;
			kpushListPo[tindex][1] = tj;
			kpushListPo[tindex][2] = i;
			continue;
		}
		if(kcost[i] - kprice[ti] + kprice[tj] - epsilon == 0&&kg[tj] > 0){
			tindex = atomicAdd(&knaCount, 1);
			kpushListNa[tindex][0] = tj;
			kpushListNa[tindex][1] = ti;
			kpushListNa[tindex][2] = i;
			continue;
		}
	}
#if FULLDEBUG
	if(threadIdx.x ==0){
		printf("get pushList\n");
	}
	__syncthreads();
#endif
	int delta,tmpi,tmpj,edgeid;
	if(threadIdx.x == 0){
		for(int i = 0; i < kpoCount; i++){
			tmpi = kpushListPo[i][0];
			tmpj = kpushListPo[i][1];
			edgeid = kpushListPo[i][2];
			delta = min(kg[tmpi], krb[edgeid] - kflow[tmpi*knumNodes + tmpj]);
			kflow[tmpi*knumNodes + tmpj] += delta;
			kg[tmpi] -= delta;
			kg[tmpj] += delta;
		}
		for(int i = 0; i < knaCount; i++){
			tmpi = kpushListNa[i][0];
			tmpj = kpushListNa[i][1];
			edgeid = kpushListNa[i][2];
			delta = min(kg[tmpi], kflow[tmpj*knumNodes + tmpi] - klb[edgeid]);
			kflow[tmpj*knumNodes + tmpi] -= delta;
			kg[tmpi] -= delta;
			kg[tmpj] += delta;
		}
	}
	__syncthreads();
#if FULLDEBUG
		if(threadIdx.x == 0){
			printf("out pushFlow\n");
		}
		__syncthreads();
#endif

	return ;
}
__device__ void priceRise(
		const int lnodes,
		const int rnodes,
		const int ledges,
		const int redges,
		const int epsilon,
		const int knumNodes,
		const int* kflow,
		const int* krb,
		const int* klb,
		const int* kprice,
		const int* kcost,
		const int* edges,
		const int* kg
		){
#if FULLDEBUG
		if(threadIdx.x == 0){
			printf("in priceRise\n");
		}
		__syncthreads();
#endif

	int ti,tj,tmpa,tmpb;
	for(int i = lnodes; i < rnodes; i++){
		if(kg[i] > 0){
			knodesRisePrice[i] = true;
		}else {
			knodesRisePrice[i] = false;
		}
	}
	__syncthreads();
	for(int i = ledges; i < redges; i++){
		ti = edges[i*2 + 0];
		tj = edges[i*2 + 1];
		if(knodesRisePrice[ti]!=knodesRisePrice[tj]){
			if(kflow[ti*knumNodes + tj] < krb[i] && !knodesRisePrice[tj]){
				tmpb = kprice[tj] + kcost[i] + epsilon - kprice[ti];
				if(tmpb >= 0){
					atomicMin(&minRise, tmpb);
				}
			}
			if(kflow[ti*knumNodes + tj] > klb[i] && knodesRisePrice[tj]){
				tmpa = kprice[ti] - kcost[i] + epsilon - kprice[tj];
				if(tmpa >= 0){
					atomicMin(&minRise, tmpa);
				}
			}
		}
	}
#if FULLDEBUG
		if(threadIdx.x == 0){
			printf("out priceRise\n");
		}
		__syncthreads();
#endif

}
__global__ void __launch_bounds__(1024, 1)
auction_algorithm_kernel(
		const int knumNodes,
		const int knumEdges,
		const int kthreadNum,
		const int kC,

		const int* kedges,
		int* kcost,
		const int* kcostRaw,
		int* kg,
		const int* kgraw,
		const int* klb,
		const int* krb,
		int* kprice,
		int* kflow){
	const int threadId = threadIdx.x;

	
	if(threadId == 0){
		printf("in kernel\n");
	}
	__syncthreads();


	int kepsilon = 1;
	int edgesDivThread;
	int edgesModThread;
	//[edgesl,edgesr) is the range of edges that the thread produre
	int ledges;
	int redges;

	int nodesDivThread;
	int nodesModThread;
	int lnodes;
	int rnodes;

	int totalIteratorNum = 0;
	int iteratorNum = 0;
	int scalingFactor = 2;
	int costScale = 9;
	int gdelta = 0;

	int kti;
	int ktj;

	edgesDivThread = knumEdges/kthreadNum;
	edgesModThread = knumEdges%kthreadNum;
	
	if(threadId < edgesModThread){
		ledges = threadId*(edgesDivThread + 1);
		redges = (threadId + 1)*(edgesDivThread + 1);
	}else {
		ledges = threadId*edgesDivThread + edgesModThread;
		redges = (threadId + 1)*edgesDivThread + edgesModThread;
	}
	
	nodesDivThread = knumNodes/kthreadNum;
	nodesModThread = knumNodes%kthreadNum;

	if(threadId < nodesModThread){
		lnodes = threadId*(nodesDivThread + 1);
		rnodes = (threadId + 1)*(nodesDivThread + 1);
	}else{
		lnodes = threadId*nodesDivThread + nodesModThread;
		rnodes = (threadId + 1)*nodesDivThread + nodesModThread;
	}
#if FULLDEBUG
	printf("threadId: %d, ledges: %d, redges: %d\n", threadId, ledges, redges);
	__syncthreads();
	for(int i = ledges; i < redges; i++){
		kflow[kedges[i*2 + 0] * knumNodes + kedges[i*2 + 1]] = atomicAdd(&justForTest, 1);
		printf("%d\n", kflow[kedges[i*2 + 0] * knumNodes + kedges[i*2 + 1]]);
	}
	__syncthreads();
#endif
	while(costScale >= 0){
#if DEBUG
		if(threadId == 0){
			printf("cost scale: \n");
		}
#endif
		for(int i = lnodes; i < rnodes; i++){
			kg[i] = kgraw[i];
		}

		int ktmp = 1<<costScale;

		for(int i = ledges; i < redges; i++){
			kti = kedges[i*2 + 0];
			ktj = kedges[i*2 + 1];
			kflow[kti * knumNodes + ktj] = 0;
			kcost[i] = kcostRaw[i]/ktmp;
		}
		for(int i = lnodes; i < rnodes; i++){
			kprice[i]*=(1 << gdelta);
		}
		__syncthreads();
		for(int i = ledges; i < redges; i++){
				kti = kedges[i*2 + 0];
				ktj = kedges[i*2 + 1];
				//todo
				if(kcost[i] - kprice[kti] + kprice[ktj] + kepsilon <= 0){
					atomicSub(kg + kti, krb[i]);
					atomicAdd(kg + ktj, krb[i]);
					kflow[kti*knumNodes+ktj] = krb[i];
				}
		}
#if FULLDEBUG
		if(threadId == 0){
			printNodes(kg, knumNodes, "kg cycle init0");
		}
		__syncthreads();
#endif
		iteratorNum = 0;
		if(threadId == 0)
		{
			kflag = true;
		}
#if FULLDEBUG
		if(threadId == 0){
			printNodes(kg, knumNodes, "g");
		}
		__syncthreads();

		for(int i = lnodes; i < rnodes; i++){
			kg[i] = 0;
		}
		kcheck(
				kg,
				lnodes,
				rnodes
			  );
		__syncthreads();
		if(threadId == 0){
			printf("\nkflag should be true: %d\n", kflag);
			kg[knumNodes/2] = 1;
			printNodes(kg, knumNodes, "g");
		}
		__syncthreads();
		kcheck(
				kg,
				lnodes,
				rnodes
			  );
		__syncthreads();
		if(threadId == 0)
			printf("\nkflag should be false: %d\n", kflag);
		break;
#endif
		__syncthreads();

		kcheck(
			kg,
			lnodes,
			rnodes
		);
		__syncthreads();
#if FULLDEBUG
		if(threadId == 0){
			printf("first check: %d\n",kflag);
			printNodes(kg, knumNodes, "kg cycle init");
		}
		__syncthreads();
#endif
		while(!kflag){
#if FULLDEBUG
			if(threadId == 0){
				printf("iteration : %d\n", iteratorNum);
			}
			__syncthreads();
#endif
			pushFlow(
				lnodes,
				rnodes,
				ledges,
				redges,
				kepsilon,
				knumNodes,
				kflow,
				krb,
				klb,
				kprice,
				kcost,
				kedges,
				kg
				);
			if(threadId == 0){
				minRise = MAXMY;
			}
			__syncthreads();
			priceRise(
				lnodes,
				rnodes,
				ledges,
				redges,
				kepsilon,
				knumNodes,
				kflow,
				krb,
				klb,
				kprice,
				kcost,
				kedges,
				kg
				);
			__syncthreads();
#if FULLDEBUG
			if(threadId == 0){
				printf("minRise: %d\n", minRise);
			}
			__syncthreads();
#endif
			if(threadId == 0){
				if(minRise == MAXMY){
					minRise = 0;
				}
			}

			__syncthreads();
			for(int i = lnodes; i < rnodes; i++){
				if(knodesRisePrice[i]){
					kprice[i] += minRise;
				}
			}
			__syncthreads();
			iteratorNum++;
			totalIteratorNum++;
			if(threadId == 0)
			{
				kflag = true;
			}
			kcheck(
				kg,
				lnodes,
				rnodes
			);
			__syncthreads();
#if FULLDEBUG
			if(threadId == 0){
				printNodes(kg, knumNodes, "kg");
				printNodes(kprice, knumNodes, "kprice");
			}
			__syncthreads();
#endif
		}

#if DEBUG
		if(threadId == 0){
			tans = 0;
		}
		__syncthreads();
		for(int i = ledges; i < redges; i++){
			kti = kedges[i*2 + 0];
			ktj = kedges[i*2 + 1];
			atomicAdd(&tans, kflow[kti*knumNodes + ktj]*kcostRaw[i]);
		}
		if(threadId == 0){
			printf("inner loop out: %d\n",iteratorNum);
			printf("temporary ans: %d\n",tans);
			printf("cost scale: %d\n", costScale);
		}
		__syncthreads();
#endif
		if(costScale ==0){
			break;
		}
		gdelta = costScale - max(0, costScale - scalingFactor);
		costScale = max(0, costScale - scalingFactor);
	}


	if(threadId == 0)
	{
		printGraph(kcost, knumNodes,"cost");
		printf("kenerl end: %d\n",totalIteratorNum);
	}
	__syncthreads();
}
hr_clock_rep timer_start, timer_mem, timer_stop;

void run_auction(
		int numNodes,
		int numEdges,
		int threadNum,
		int dC,

		int* hedges,
		int* hcost,
		int* hg,
		int* hlb,
		int* hrb,

		int* hflow){
	cout << "start run_auction\n";
	int* dedges;
	int* dcost;
	int* dcostRaw;
	int* dg;
	int* dgraw;
	int* dlb;
	int* drb;

	int* dprice;

	int* dflow;

	timer_start = get_globaltime();
	hipMalloc((void **)&dedges, EDGESIZE*2*sizeof(int));
	hipMalloc((void **)&dcost, EDGESIZE*sizeof(int));
	hipMalloc((void **)&dcostRaw, EDGESIZE*sizeof(int));

	hipMalloc((void **)&dg, SIZE*sizeof(int));
	hipMalloc((void **)&dgraw, SIZE*sizeof(int));
	hipMalloc((void **)&dlb, EDGESIZE*sizeof(int));
	hipMalloc((void **)&drb, EDGESIZE*sizeof(int));


	hipMalloc((void **)&dprice, SIZE*sizeof(int));

	hipMalloc((void **)&dflow, SIZE*SIZE*sizeof(int));


	hipMemcpy(dedges, hedges, EDGESIZE*2*sizeof(int), hipMemcpyHostToDevice);
	
	hipMemcpy(dcost, hcost, EDGESIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dcostRaw, hcost, EDGESIZE*sizeof(int), hipMemcpyHostToDevice);


	hipMemcpy(dg, hg, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dgraw, hg, SIZE*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(dlb, hlb, EDGESIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(drb, hrb, EDGESIZE*sizeof(int), hipMemcpyHostToDevice);


	timer_mem = get_globaltime();
	hipProfilerStart();
	cout << "start kernel\n";
	auction_algorithm_kernel<<<1,threadNum>>>
		(
		numNodes,
		numEdges,
		threadNum,
		dC,
		dedges,
		dcost,
		dcostRaw,
		dg,
		dgraw,
		dlb,
		drb,
		dprice,
		dflow);
	hipProfilerStop();
	hipDeviceSynchronize();
	timer_stop = get_globaltime();
	hipMemcpy(hflow, dflow, SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost);
	
	int ans = 0;
//	for(int i = 0; i < numNodes; i++){
//		for(int j = 0; j < numNodes; j++){
//			ans += hflow[i*numNodes + j]*hcost[i*numNodes+ j];
//			cout << hflow[i*numNodes + j] << " ";
//		}
//	}
	cout << "ans:  " << ans << endl;

}


void initmy(
		int *dc,
		int *edges,
		int *cost,
		int *hg,
		int *lb,
		int *rb
		){
	cout << "start read in graph..\n";
	int tnumNodes;
	int tCapacity = 0;
	int tmaxCost = 0;
	cin >> tnumNodes;
	cout << "tnumNodes: "<< tnumNodes << endl;
	memset(cost, MAXMY, sizeof(cost));
	memset(edges, 0, sizeof(edges));
	memset(hg, 0, sizeof(hg));
	char a;
	int fid;
	int aNUm;
	cin >> aNUm;
//	cout << "aNUm " << aNUm << endl;
	for(int i = 0; i < aNUm; i++){
		cin >> a >> fid;
		cin >> hg[fid-1];
//		cout << a << " " << fid << " " << g[fid-1] << endl;
	}
	int ti,tj;
	int edgeNum = 0;
	while(true){
		cin >> a >> ti >> tj;
		if(ti == tj&&ti==0){
			break;
		}
		ti--;tj--;
		edges[edgeNum*2] = ti;
		edges[edgeNum*2 + 1] = tj;

		cin >> lb[edgeNum] >> rb[edgeNum] >>  cost[edgeNum] ;
//		cout << a << "\t" << ti << " " << tj << " " << cost[ti*SIZE + tj] <<" " << lb[ti*SIZE + tj] << " " << rb[ti*SIZE + tj] <<  endl;
//		cost[ti][tj] *= nodeNum;
//		cost[ti][tj] %= 4000;
		tmaxCost = max(cost[edgeNum], tmaxCost);
		tCapacity = max(rb[edgeNum], tCapacity);
		edgeNum++;
	}
	cout << "EDGENUM: " << edgeNum << endl;
	*dc = tmaxCost;
	/*
	for(int i = 0; i < edgeNum;i ++){
		printf("%d\t",cost[edges[(i * 2 + 0)]*SIZE + edges[i * 2 + 1]]);
	}
	cout << endl;
	for(int i = 0; i < edgeNum; i++){
		printf("%d\t",costa[i]);
	}
	*/
	cout << "read end\n";
}

int main(int argc, char *argv[]){
	int threadNum = 1024;
	int numNodes = SIZE;
	int numEdges = EDGESIZE;
	int hC;
	int *hedges = new int[EDGESIZE*2];
	int *hcost = new int[EDGESIZE];
	int *hg = new int[SIZE];
	int *hlb = new int[EDGESIZE];
	int *hrb = new int[EDGESIZE];

	int *hflow = new int[SIZE*SIZE];
	int *hflowa = new int[EDGESIZE];
	memset(hflow, 0, sizeof(hflow));

	initmy(
		&hC,
		hedges,
		hcost,
		hg,
		hlb,
		hrb
	);

	run_auction(
		numNodes,
		numEdges,
		threadNum,
		hC,

		hedges,
		hcost,
		hg,
		hlb,
		hrb,

		hflow
	);

	std::cerr << "run_acution takes "<< (timer_stop - timer_start)*get_timer_period() << "ms totally.\n";
	std::cerr << "memory copy takes "<< (timer_mem - timer_start)*get_timer_period() << "ms totally.\n";
	std::cerr << "kernel takes "<< (timer_stop - timer_mem)*get_timer_period() << "ms totally.\n";
	return 0;
}
