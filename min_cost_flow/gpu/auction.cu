#include "hip/hip_runtime.h"
#include "auction.cuh"
#include "min_cost_flow_gpu.h"

#define MAXMY 0x3f3f3f3f
#define MAXCOST 10000
namespace mcfgpu {
struct PushEdge
{
	int edge;
	int delta;
	int gId; ///< Id of nodes which's grow decrease;
	int nextEdge;
	bool direct;
};
struct PushNode
{
	int nodeId;
	int firstEdgeId;
};



template<class GRA, class NM, class AM, class NI, class AI>
struct AuctionState
{
	struct PushEdge* kpushList;

	int* kpushListFlag;
    bool* knodesRisePrice; ///< length of #nodes 

    void initialize(Graph<GRA, NM, AM, NI, AI> const& G)
    {
        printf("initialize state with %d nodes\n", G.getNodesNum());
		kpushList = nullptr;
        knodesRisePrice = nullptr; 

		hipError_t status = hipMalloc((void **)&kpushList, G.getEdgesNum()*sizeof(PushEdge));
        if (status != hipSuccess) 
        { 
            printf("hipMalloc failed for kpushList\n"); 
        } 

		status = hipMalloc((void **)&kpushListFlag, G.getNodesNum()*sizeof(int));
        if (status != hipSuccess) 
        { 
            printf("hipMalloc failed for kpushListFlag\n"); 
        } 
		hipMemset(kpushListFlag, 0, G.getNodesNum()*sizeof(int));
        status = hipMalloc((void **)&knodesRisePrice, G.getNodesNum()*sizeof(bool));
        if (status != hipSuccess) 
        { 
            printf("hipMalloc failed for knodesRisePrice\n"); 
        }
        printf("end init\n");
    }

    void destroy()
    {
        hipFree(knodesRisePrice);
		hipFree(kpushList);
    }
};

template<class GRA, class NM, class AM, class NI, class AI>
//pushlist is not good
__device__ void pushFlow(
		Graph<GRA, NM, AM, NI, AI> &G,
        AuctionState<GRA, NM, AM,NI, AI>& state, 
		const int lnodes,
		const int rnodes,
        const int node_step, 
		const int ledges,
		const int redges,
        const int edge_step, 
		const int epsilon,
		const int knumNodes, 
		int& kpushCount,
		int& kpushFlag
		){
#if FULLDEBUG
	if(threadIdx.x ==0){
		printf("in pushFlow\n");
	}
	__syncthreads();
#endif
	if(threadIdx.x ==0){
		kpushCount = 0;
	}
	__syncthreads();

	for(int i = ledges; i < redges; i += edge_step){
		int ti,tj,mindex;
        auto const& edge = G.edge(i); 
		ti = edge.source;
		tj = edge.sink;
        int value = G.atCost(i) - G.atPrice(ti) + G.atPrice(tj);
		if(value + epsilon == 0 && G.atGrow(ti) >0){
			mindex = atomicAdd(&kpushCount, 1);
			state.kpushList[mindex].edge = i;
			state.kpushList[mindex].direct = true;
			state.kpushList[mindex].gId = ti;
			state.kpushList[mindex].delta = G.atRb(i) - G.atFlow(i);
		}
		else if (value - epsilon == 0 && G.atGrow(tj) > 0){
			mindex = atomicAdd(&kpushCount, 1);
			state.kpushList[mindex].edge = i;
			state.kpushList[mindex].direct = false;
			state.kpushList[mindex].gId = tj;
			state.kpushList[mindex].delta = G.atFlow(i) - G.atLb(i);
		}
	}
#if (DEBUG && DEBUG1)
	if(threadIdx.x ==0){
		printf("get pushList\n");
		if(kpushCount == 0){
			printf("no edge to push!!\n");
		}
		for(int i = 0; i < kpushCount; i++){
			printf("%d\t", state.kpushList[i].edge);
		}
		printf("\n");
	}
	__syncthreads();
#endif
	__syncthreads();
	int delta,tmpi,tmpj,tmpk;
	/*
	int tdivid = kpushCount / blockDim.x;
	int tmod = kpushCount % blockDim.x;
	int tlb,trb;
	if(threadIdx.x < tmod){
		tlb = threadIdx.x * (tdivid + 1);
		trb = (threadIdx.x + 1) * (tdivid + 1);
	}else{
		tlb = threadIdx.x * tdivid + tmod;
		trb = (threadIdx.x + 1)*tdivid + tmod;
	}
	
	do{
		__syncthreads();
		if(threadIdx.x == 0){
			kpushFlag = 0;
		}
		__syncthreads();
		for(int i = tlb; i < trb; i++){
			tmpk = state.kpushList[i].edge;
			tmpi = state.kpushList[i].gId;
			auto const& edge = G.edge(tmpk);
			if(state.kpushList[i].delta != 0){
				if(atomicAdd(&state.kpushListFlag[tmpi], 1) == 0){
					if(state.kpushList[i].direct){
						tmpj = edge.sink;
						delta = min(G.atGrow(tmpi), state.kpushList[i].delta);
						G.setFlow(tmpk, G.atFlow(tmpk) + delta);
					}else{
						tmpj = edge.source;
						delta = min(G.atGrow(tmpi), state.kpushList[i].delta);
						G.setFlow(tmpk, G.atFlow(tmpk) - delta);
					}
					state.kpushList[i].delta -= delta;
					G.atomicSubGrow(tmpi, delta);
					G.atomicAddGrow(tmpj, delta);
					if(delta != 0){
						atomicAdd(&kpushFlag,1);
					}
				}
				atomicSub(&state.kpushListFlag[tmpi], 1);
			}
		}
		__syncthreads();
	}while(kpushFlag != 0);
*/
	if(threadIdx.x == 0){
		for(int i = 0; i < kpushCount; i++){
			tmpk = state.kpushList[i].edge;
			tmpi = state.kpushList[i].gId;
//				printf("get in: %d\n", tmpi);
			auto const& edge = G.edge(tmpk);
			if(state.kpushList[i].direct){
				tmpj = edge.sink;
				delta = min(G.atGrow(tmpi), state.kpushList[i].delta);
				G.setFlow(tmpk, G.atFlow(tmpk) + delta);
			}else{
				tmpj = edge.source;
				delta = min(G.atGrow(tmpi), state.kpushList[i].delta);
				G.setFlow(tmpk, G.atFlow(tmpk) - delta);
			}
			state.kpushList[i].delta -= delta;
			G.atomicSubGrow(tmpi, delta);
			G.atomicAddGrow(tmpj, delta);
		}
	}
    
	__syncthreads();
#if FULLDEBUG
		if(threadIdx.x == 0){
			printf("out pushFlow\n");
		}
		__syncthreads();
#endif

	return ;
}


template<class GRA, class NM, class AM, class NI, class AI>
__device__ void priceRise(
		Graph<GRA, NM, AM, NI, AI> &G,
        AuctionState<GRA, NM, AM, NI, AI>& state, 
		const int lnodes,
		const int rnodes,
        const int node_step, 
		const int ledges,
		const int redges,
        const int edge_step, 
		const int epsilon,
		const int knumNodes, 
        int& minRise
		){
#if DEBUG
		if(threadIdx.x == 0){
			printf("in priceRise\n");
		}
		__syncthreads();
#endif

	int ti,tj,tmpa,tmpb;
	for(int i = lnodes; i < rnodes; i += node_step){
		if(G.atGrow(i) > 0){
			state.knodesRisePrice[i] = true;
		}else {
			state.knodesRisePrice[i] = false;
		}
	}
	__syncthreads();
	for(int i = ledges; i < redges; i += edge_step){
        auto const& edge = G.edge(i);
		ti = edge.source;
		tj = edge.sink;
		if(state.knodesRisePrice[ti] != state.knodesRisePrice[tj]){
#if DEBUG
					if(tmpb == 0){
						printf("possible\n");
					}
#endif

			if(G.atFlow(i) < G.atRb(i) && state.knodesRisePrice[ti]){
				tmpb = G.atPrice(tj) + G.atCost(i) + epsilon - G.atPrice(ti);
				if(tmpb >= 0){
#if DEBUG
					if(tmpb == 0){
						printf("miRise == 0: node: %d, price: %d, Cost: %d,po\n ",ti, G.atPrice(ti),G.atCost(i));
					}
#endif
					atomicMin(&minRise, tmpb);
				}
			}
			if(G.atFlow(i) > G.atLb(i) && state.knodesRisePrice[tj]){
				tmpa = G.atPrice(ti) - G.atCost(i) + epsilon - G.atPrice(tj);
				if(tmpa >= 0){
#if DEBUG
					if(tmpb == 0){
						printf("miRise == 0: node: %d, price: %d, Cost: %d,po\n ",ti, G.atPrice(ti),G.atCost(i));
					}
#endif
					atomicMin(&minRise, tmpa);
				}
			}
		}
	}
#if DEBUG
		__syncthreads();
		if(threadIdx.x == 0){
			printf("out priceRise\n minRise = %d\n",minRise);
		}
		__syncthreads();
#endif
	__syncthreads();

}

template<class GRA, class NM, class AM, class NI, class AI>
__global__ void __launch_bounds__(1024)
auction_algorithm_kernel(
		Graph<GRA, NM, AM, NI, AI> G, 
        AuctionState<GRA, NM, AM, NI, AI> state,
        int *dTotalCost,
        int *dflow,
        int *dprice
){
    __shared__ int kepsilon;
	__shared__ int totalIteratorNum;
	__shared__ int iteratorNum;
	__shared__ int scalingFactor;
	__shared__ int costScale;
	__shared__ int gdelta;
	__shared__ int knumNodes;
	__shared__ int knumEdges;
	__shared__ int edgesDivThread;
	__shared__ int nodesDivThread;
    __shared__ int kflag; 
    __shared__ int minRise;
	__shared__ int kpushCount;
	__shared__ int kpushFlag;
    __shared__ int tans;

	const int threadId = threadIdx.x;


    if (threadId == 0) {
        kepsilon = 1; 
        totalIteratorNum = 0; 
        iteratorNum = 0; 
        scalingFactor = 2; 
        costScale = 9; 
        gdelta = 0; 
        knumNodes = G.getNodesNum();
        knumEdges = G.getEdgesNum();
        edgesDivThread = max(knumEdges / blockDim.x, 1);
        nodesDivThread = max(knumNodes / blockDim.x, 1);

		printf("in kernel\n");
    }
    __syncthreads();
    
	//[edgesl,edgesr) is the range of edges that the thread produre
	const int ledges = threadId * edgesDivThread;
	const int redges = min(ledges + edgesDivThread, knumEdges);
    const int edge_step = 1; 

	const int lnodes = threadId * nodesDivThread;
	const int rnodes = min(lnodes + nodesDivThread, knumNodes);
    const int node_step = 1; 

	int kti;
	int ktj;
	while(costScale >= 0){
#if DEBUG
		if(threadId == 0){
			printf("cost scale: %d\n",costScale);
		}
#endif
		for(int i = lnodes; i < rnodes; i += node_step){
			G.setGrow(i , G.atGrowRaw(i));
		}

		int ktmp = 1<<costScale;
		for(int i = ledges; i < redges; i += edge_step){
			G.setFlow(i, 0);
			if(G.atCostRaw(i) <= G.getMaxCost()){
				G.setCost(i, G.atCostRaw(i)/ktmp);
			}
		}

		for(int i = lnodes; i < rnodes; i++){
			G.setPrice(i, G.atPrice(i)*(1 << gdelta));
		}
		__syncthreads();
		for(int i = ledges; i < redges; i += edge_step){
            auto const& edge = G.edge(i);
			kti = edge.source;
			ktj = edge.sink;
			if(G.atCost(i) - G.atPrice(kti) + G.atPrice(ktj) + kepsilon <= 0){
				G.atomicSubGrow(kti, G.atRb(i));
				G.atomicAddGrow(ktj, G.atRb(i));
				G.setFlow(i, G.atRb(i));
			}
		}
		if(threadId == 0)
		{
            iteratorNum = 0;
			kflag = true;
		}
		__syncthreads();

		for(int i = lnodes; i < rnodes; i += node_step){
			if(G.atGrow(i) != 0){
				atomicAnd(&kflag, 0);
			}
		}
		__syncthreads();
		while(!kflag){
#if FULLDEBUG
			if(threadId == 0){
				printf("iteration : %d\n", iteratorNum);
			}
			__syncthreads();
#endif
            pushFlow(
                    G,
                    state, 
                    lnodes,
                    rnodes,
                    node_step, 
                    ledges,
                    redges,
                    edge_step, 
                    kepsilon,
                    knumNodes, 
					kpushCount,
					kpushFlag
                    );
			if(threadId == 0){
				minRise = MAXMY;
			}

			__syncthreads();
            priceRise(
                    G,
                    state, 
                    lnodes,
                    rnodes,
                    node_step, 
                    ledges,
                    redges,
                    edge_step, 
                    kepsilon,
                    knumNodes, 
                    minRise
                    );
			__syncthreads();
			if(threadId == 0){
				if(minRise == MAXMY){
					minRise = 0;
				}
			}

			__syncthreads();
			for(int i = lnodes; i < rnodes; i += node_step){
				if(state.knodesRisePrice[i]){
					G.setPrice(i, G.atPrice(i) + minRise);
				}
			}
			__syncthreads();
			if(threadId == 0)
			{
                iteratorNum++;
                totalIteratorNum++;
				kflag = true;
			}
			for(int i = lnodes; i < rnodes; i += node_step){
				if(G.atGrow(i) != 0){
					atomicAnd(&kflag, 0);
				}
			}
			__syncthreads();
#if (DEBUG&&DEBUG1)
			if(threadId == 0){

//				G.printGrow();
				if(minRise == 0)
					printf("iteration : %d  minRise: %d\n", iteratorNum ,minRise);
				int unfeed = 0;
				for(int i = 0; i < G.getNodesNum(); i++){
					if(G.atGrow(i) > 0){
						unfeed += G.atGrow(i);
					}
				}
				printf("unfeed source is %d\n", unfeed);
			}
			__syncthreads();
#endif
		}
#if DEBUG
		if(threadId == 0){
			tans = 0;
		}
		__syncthreads();
		for(int i = ledges; i < redges; i += edge_step){
			atomicAdd(&tans, G.atFlow(i)*G.atCostRaw(i));
		}
		if(threadId == 0){
			printf("inner loop out\n");
			printf("temporary ans: %d\n",tans);
			printf("cost scale: %d\n", costScale);
			printf("iteratorNum: %d\n", iteratorNum);
		}
		__syncthreads();

#endif
		if(costScale ==0){
			break;
		}
        if (threadId == 0) {
            gdelta = costScale - max(0, costScale - scalingFactor);
            costScale = max(0, costScale - scalingFactor);
        }
        __syncthreads();
	}

	if(threadId == 0)
	{
		tans = 0;
	}
    
	for(int i = ledges; i < redges; i += edge_step){
		atomicAdd(&tans, G.atFlow(i)*G.atCostRaw(i));
	}

	if(threadId == 0){
		printf("inner loop out\n");
		printf("temporary ans: %d\n",tans);
		printf("cost scale: %d\n", costScale);
		printf("iteratorNum: %d\n", iteratorNum);
		printf("totalIteratorNum: %d\n", totalIteratorNum);
		printf("kenerl end\n");

        *dTotalCost = tans;
        for (int i = 0; i < G.getEdgesNum(); ++i) {
            dflow[i] = G.atFlow(i);
        }
        int totalPrice = 0;
        for (int i = 0; i < G.getNodesNum(); ++i) {
            dprice[i] = G.atPrice(i);
            totalPrice += G.atPrice(i);
            //printf("[%d] : %d\n", i, G.atPrice(i));
        }

	}

	__syncthreads();
}


hr_clock_rep timer_start, timer_mem, timer_stop;
template<class G, class NM, class AM, class NI, class AI>
void GPU<G, NM, AM, NI, AI>::run_auction(Graph<G, NM, AM, NI, AI> auctionGraph, int threadNum){
	std::cout << "start run_auction\n";
    ProblemType status = ProblemType::INFEASIBLE;
	hipProfilerStart();
	std::cout << "start kernel\n";
    
    size_t totalCostSize = sizeof(int);
    size_t flowSize = sizeof(int ) * numEdges;
    size_t priceSize = sizeof(int) * numNodes;

    int *hTotalCost, *dTotalCost;
    hTotalCost = (int *)malloc(totalCostSize);
    hipMalloc((void **)&dTotalCost, totalCostSize);
    hipMemcpy(dTotalCost, hTotalCost, totalCostSize, hipMemcpyHostToDevice);

    int *hflow, *dflow;
    hflow = (int *)malloc(flowSize);
    hipMalloc((void **)&dflow, flowSize);
    hipMemcpy(dflow, hflow, flowSize, hipMemcpyHostToDevice);

    int *hprice, *dprice;
    hprice = (int *)malloc(priceSize);
    hipMalloc(&dprice, priceSize);
	hipMemcpy(dprice, hprice, priceSize, hipMemcpyHostToDevice);


    AuctionState<G, NM, AM, NI, AI> state; 
    state.initialize(auctionGraph);
    auction_algorithm_kernel<<<1,threadNum>>>
		(
		auctionGraph, 
      state, 
      dTotalCost,
       dflow,
      dprice
	);
    status = ProblemType::OPTIMAL;
    state.destroy();
	hipProfilerStop();
	hipDeviceSynchronize();
	timer_stop = get_globaltime();
    
    hipMemcpy(hTotalCost, dTotalCost, totalCostSize, hipMemcpyDeviceToHost);
    hipMemcpy(hflow, dflow, flowSize, hipMemcpyDeviceToHost); 
    hipMemcpy(hprice, dprice, priceSize, hipMemcpyDeviceToHost);
    
    memcpy(res.flowMap, hflow, flowSize);
    memcpy(res.potential, hprice, priceSize);
    res.totalCost = *hTotalCost;
    res.pt = status;
    

  /* if (res.totalCost > MAXCOST) {
        printf("totalcost is too big: %d\n", res.totalCost);
        result = ProblemType::UNBOUNDED;
    }
*/
    hipFree(dTotalCost);
    hipFree(dflow);
    hipFree(dprice);
    free(hTotalCost);
    free(hflow);
    free(hprice);

    return ;

}

template<class G, class NM, class AM, class NI, class AI>
void GPU<G, NM, AM, NI, AI>::run(){
	int threadNum = 1024;
//	initmy(&hC,hedges,hcost,hg,hlb,hrb	);
	timer_start = get_globaltime();
	Graph<G, NM, AM, NI, AI> auctionGraph(Graph<G, NM, AM, NI, AI>::edgeList, _map, _supply, _capacity_upper, _cost);
	timer_mem = get_globaltime();
//	Graph auctionGraph = Graph(Graph::matrix,numNodes, numEdges, hC, hedges, hcost, hlb, hrb, hg);

	run_auction(
		auctionGraph,
		threadNum
	);

    std::cout << "run_acution takes "<< (timer_stop - timer_start)*get_timer_period() << "ms totally.\n";
	std::cout << "memory copy takes "<< (timer_mem - timer_start)*get_timer_period() << "ms totally.\n";
	std::cout << "kernel takes "<< (timer_stop - timer_mem)*get_timer_period() << "ms totally.\n";

	std::cerr << "run_acution takes "<< (timer_stop - timer_start)*get_timer_period() << "ms totally.\n";
	std::cerr << "memory copy takes "<< (timer_mem - timer_start)*get_timer_period() << "ms totally.\n";
	std::cerr << "kernel takes "<< (timer_stop - timer_mem)*get_timer_period() << "ms totally.\n";
	return ;
} 

};//end namespace mcfgpu
